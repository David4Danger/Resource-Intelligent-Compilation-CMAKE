/*
 * CUDA for Prognostics - PRNG Implementation
 *
 * Created by: David Skudra (david.skudra@nasa.gov)
 * Created on: Jan 23rd, 2018
*/

#include <iostream>
#include "hiprand.h"
#include "prng.h"
#include "debugCFP.h"

void prngUniformDouble (double *devStart, int count) {
  int status;
  hiprandGenerator_t gen;

  status = hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_MRG32K3A);
  DEBUG(DB_CURAND, "Created a PRNG for uniformly distributed doubles.");
  status |= hiprandSetPseudoRandomGeneratorSeed(gen, 4294967296ULL^time(NULL));
  status |= hiprandGenerateUniformDouble(gen, devStart, count);
  status |= hiprandDestroyGenerator(gen);
  if (status != HIPRAND_STATUS_SUCCESS) {
    std::cout << "CuRand Failure!" << std::endl;
    exit(EXIT_FAILURE);
  }
  DEBUG(DB_CURAND, "Successfully generated doubles.");
}

void prngNormalDouble (double *devStart, int count, double mean, double stddev) {
  int status;
  hiprandGenerator_t gen;

  status = hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_MRG32K3A);
  DEBUG(DB_CURAND, "Created a PRNG for normally distributed doubles.");
  status |= hiprandSetPseudoRandomGeneratorSeed(gen, 4294967296ULL^time(NULL));
  status |= hiprandGenerateNormalDouble(gen, devStart, count, mean, stddev);
  status |= hiprandDestroyGenerator(gen);
  if (status != HIPRAND_STATUS_SUCCESS) {
    std::cout << "CuRand Failure!" << std::endl;
    exit(EXIT_FAILURE);
  }
  DEBUG(DB_CURAND, "Successfully generated doubles.");
}
